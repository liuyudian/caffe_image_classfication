/************************************************************************/
/* This source code is	free for both academic and industry use.         */
/* Some important information for better using the source code could be */
/* found in the project page: http://mmcheng.net/bing					*/
/************************************************************************/

#include "stdafx.h"
#include "Objectness.h"
#include "ValStructVec.h"
#include "CmShow.h"


// Uncomment line line 14 in Objectness.cpp to remove counting times of image reading.

void RunObjectness(CStr &resName, double base, int W, int NSS, int numPerSz, CStr &dataPath);

int main(int argc, char* argv[])
{
	//DataSetVOC::importImageNetBenchMark();
	//DataSetVOC::cvt2OpenCVYml("C:/WkDir/DetectionProposals/VOC2007/Annotations/");
	if(argc < 2){
		std::cerr << "Please pass the data path to as first argument" << std::endl;
		return 1;
	}

	//datapath:"/home/chigo/working/research/Bing/Objectness-master/IN2016_Test/"
	//datapath:"/home/chigo/working/research/Bing/Objectness-master/VOC2007/"
	CStr dataPath = argv[1];
	RunObjectness("WinRecall.m", 2, 8, 2, 130, dataPath);

	//Predict
	//Predict("WinRecall.m", 2, 8, 2, 130, dataPath);
	
	return 0;
}

void RunObjectness(CStr &resName, double base, int W, int NSS, int numPerSz, CStr &dataPath)
{
	srand(131);//srand((unsigned int)time(NULL));
	//omp_set_num_threads(16);
	DataSetVOC voc2007(dataPath); 
	voc2007.loadAnnotations();
	//voc2007.loadDataGenericOverCls();

	cout << "Dataset:'" << _S(voc2007.wkDir) << "' with " << voc2007.trainNum << " training and " << voc2007.testNum << " testing" << endl;
	cout << _S(resName) << " Base = " << base << ", W = " << W << ", NSS = " << NSS << ", perSz = " << numPerSz << endl;

	printf("Objectness Init...\n");
	Objectness objNess(voc2007, base, W, NSS);
	printf("Objectness Init end!!\n");
    
	vector<vector<Vec4i>> boxes;
	//objNess.getObjBndBoxesForTests(boxes, 250);
	objNess.getObjBndBoxesForTestFast(boxes, numPerSz);
	//objNess.getObjBndBoxesForTrainsEva(boxes, numPerSz);
	//objNess.getRandomBoxes(boxes);
	//objNess.evaluatePerClassRecall(boxes, resName, 2000);
	//objNess.illuTestReults(boxes);
}

/*
void Predict(CStr &resName, double base, int W, int NSS, int numPerSz, CStr &dataPath)
{
    srand((unsigned int)time(NULL));
    DataSetVOC voc2007(dataPath);

    Objectness objNess(voc2007, base, W, NSS);

    vector<vector<Vec4i>> boxesTests;

	string inputfile = "/home/chigo/image/test/test0313_1w/7387.jpg";
    objNess.Predict( inputfile, numPerSz );
}
*/

